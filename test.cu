#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>


#include "error.cuh"


#define BX blockIdx.x
#define BY blockIdx.y
#define TX threadIdx.x
#define TY threadIdx.y
#define DX blockDim.x 
#define DY blockDim.y
#define TIDX blockDim.x * blockIdx.x + threadIdx.x
#define TIDY blockDim.y * blockIdx.y + threadIdx.y


template<typename T> __global__ void mm_kernel(const T * a, const T * b, T * c, const int ra, const int ca, const int cb);


template<typename T>
void mm_device(const T * a, const T * b, T * c, const int ra, const int ca, const int cb) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  T * ad; T * bd; T * cd;
  size_t sz_a = ra * ca * sizeof(T);
  size_t sz_b = ca * cb * sizeof(T);
  size_t sz_c = ra * cb * sizeof(T);

  /*
  gpuCheckErr(hipMalloc((void**)&ad, sz_a));
  gpuCheckErr(hipMalloc((void**)&bd, sz_b));
  gpuCheckErr(hipMalloc((void**)&cd, sz_c));

  gpuCheckErr(hipMemcpy(ad, a, sz_a, hipMemcpyHostToDevice));
  gpuCheckErr(hipMemcpy(bd, b, sz_b, hipMemcpyHostToDevice));
  gpuCheckErr(hipMemcpy(cd, c, sz_c, hipMemcpyHostToDevice));
  */
  dim3 Threads(16, 16);
  int bx = (cb + Threads.x - 1) / Threads.x; bx = bx < 1024 ? bx : 1024;
  int by = (ra + Threads.y - 1) / Threads.y; by = by < 1024 ? by : 1024;		
  dim3 Grid(bx, by, 1);
  
  //printf("..dbg a(%d, %d), b(%d, %d), grid(%d,%d), threads(%d,%d)\n", ra, ca, ca, cb, bx, by, Threads.x, Threads.y);
  
  hipEventRecord(start);
  mm_kernel<T><<<Grid, Threads>>>(ad, bd, cd, ra, ca, cb);
  gpu_check_err(hipPeekAtLastError());
  gpu_check_err(hipDeviceSynchronize());
  hipEventRecord(stop);
  
  hipMemcpy(c, cd, sz_c, hipMemcpyDeviceToHost);
  hipFree(ad);
  hipFree(bd);
  hipFree(cd);
  
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  printf("..gpu_mm(%3.1fms)\n", ms);
}

template<typename T> __global__ void mm_kernel(const T * a, const T * b, T * c, const int ra, const int ca, const int cb) {
  // note : assumed that thread indices cover matrix 
  int tx = TIDX; // col
  int ty = TIDY; // row
  
  if (tx >= cb || ty >= ra) return;
  
  const int r_ca = ca - ca / DX * DX;  
  int num_mults = ca / DX;  
  int mm = (r_ca > 0 ? num_mults + 1 : num_mults);  
  int cidx = ty * cb + tx;
  
  for (int i = 0; i < mm; ++i) {
    int sa = DY * (i + ca * BY); // move to "right" in matrix "A" by 16x16 chunks 
    int sb = DX * (i * cb + BX); // move "down" matrix B by 16x16 chunks
    
    const T * sm_a = &(a[sa]); // collect sub-matrix of A
    const T * sm_b = &(b[sb]); // collect sub-matrix of B
    
    // fill one element of result matrix "c" 
    int mx = i >= num_mults ? r_ca : DX;    
    int cc = ca * TY;
    
    for (int j = 0; j < mx; ++j) {
      c[cidx] += sm_a[cc + j] * sm_b[cb * j + TX];
    }
    //__syncthreads();
  }  
}

//----------------------------------
// template specializations
//----------------------------------
template void mm_device(const char * a, const char * b, char * c, const int ra, const int ca, const int cb);
template void mm_device(const int * a, const int * b, int * c, const int ra, const int ca, const int cb);
template void mm_device(const float * a, const float * b, float * c, const int ra, const int ca, const int cb);
template void mm_device(const double * a, const double * b, double * c, const int ra, const int ca, const int cb);

template __global__ void mm_kernel(const char * a, const char * b, char * c, const int ra, const int ca, const int cb);
template __global__ void mm_kernel(const int * a, const int * b, int * c, const int ra, const int ca, const int cb);
template __global__ void mm_kernel(const float * a, const float * b, float * c, const int ra, const int ca, const int cb);
template __global__ void mm_kernel(const double * a, const double * b, double * c, const int ra, const int ca, const int cb);
